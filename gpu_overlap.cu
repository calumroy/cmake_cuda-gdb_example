// Include stdlib.h
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
// #include <helper_functions.h>
// #include <helper_cuda.h>

std::vector<int> flattenVector(const std::vector<std::vector<int>> &vec2D)
{
    std::vector<int> vec1D;
    for (const auto &vec : vec2D)
    {
        vec1D.insert(vec1D.end(), vec.begin(), vec.end());
    }
    return vec1D;
}

std::vector<int> flattenVector(const std::vector<std::vector<std::vector<std::vector<int>>>> &vec4D)
{
    std::vector<int> vec1D;
    for (const auto &vec3D : vec4D)
    {
        for (const auto &vec2D : vec3D)
        {
            for (const auto &vec : vec2D)
            {
                vec1D.insert(vec1D.end(), vec.begin(), vec.end());
            }
        }
    }
    return vec1D;
}

std::vector<std::vector<int>> unflattenVector(const std::vector<int> &vec1D, size_t numRows, size_t numCols)
{
    std::vector<std::vector<int>> vec2D(numRows, std::vector<int>(numCols));
    size_t index = 0;
    for (size_t i = 0; i < numRows; i++)
    {
        for (size_t j = 0; j < numCols; j++)
        {
            vec2D[i][j] = vec1D[index];
            index++;
        }
    }
    return vec2D;
}

std::vector<std::vector<std::vector<std::vector<int>>>> unflattenVector(const std::vector<int> &vec1D, size_t numLayers, size_t numChannels, size_t numRows, size_t numCols)
{
    std::vector<std::vector<std::vector<std::vector<int>>>> vec4D(numLayers, std::vector<std::vector<std::vector<int>>>(numChannels, std::vector<std::vector<int>>(numRows, std::vector<int>(numCols))));
    size_t index = 0;
    for (size_t l = 0; l < numLayers; l++)
    {
        for (size_t c = 0; c < numChannels; c++)
        {
            for (size_t i = 0; i < numRows; i++)
            {
                for (size_t j = 0; j < numCols; j++)
                {
                    vec4D[l][c][i][j] = vec1D[index];
                    index++;
                }
            }
        }
    }
    return vec4D;
}

///-----------------------------------------------------------------------------
///
/// sliding_window_kernel      A kernel function that performs a sliding window operation on a matrix.
///                            This kernel function oerates on a simualted 2D matrix, but the matrix is
///                            actually stored as a 1D array. The kernel function is designed to be
///                            launched with a 2D grid of 2D blocks. Each thread in the block will
///                            perform the sliding window operation on a single element in the input
///                            matrix. The output matrix will also be a 1D vector simulating a 4D vector with dimensions
///                            rows x cols x neigh_rows x neigh_cols.
///                            Each element at the output[i * cols + j] will be a 2D matrix (simulated by a flattened 1D vector)
///                            containing the neighbourhood of the input matrix element input[i * cols + j].
///
/// @param[in] input           A pointer to the input matrix on the GPU.
/// @param[out] output         A pointer to the output matrix on the GPU.
/// @param[in] rows            The number of rows in the input matrix.
/// @param[in] cols            The number of columns in the input matrix.
/// @param[in] neib_rows       The number of rows in the neighbourhood.
/// @param[in] neib_cols       The number of columns in the neighbourhood.
/// @param[in] step_rows       The number of rows to step the neighbourhood for each iteration.
/// @param[in] step_cols       The number of columns to step the neighbourhood for each iteration.
/// @param[in] wrap_mode       A flag indicating whether the neighbourhood should wrap around the input matrix.
/// @param[in] center_neigh    A flag indicating whether the neighbourhood should be centered over the current element in the input matrix.
///-----------------------------------------------------------------------------
__global__ void sliding_window_kernel(int *input, int *output, int rows, int cols, int neib_rows, int neib_cols, int step_rows, int step_cols, bool wrap_mode, bool center_neigh)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Row index of the thread index
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Column index of the thread index

    // The threads in the block that are outside the bounds of the input matrix do nothing.
    if (i < rows && j < cols)
    {
        for (int ii = 0; ii < neib_rows; ++ii)
        {
            for (int jj = 0; jj < neib_cols; ++jj)
            {
                int x = i + ii * step_rows;
                int y = j + jj * step_cols;

                // If the "center_neigh" flag is set, center the neighbourhood over the current element in the input matrix.
                if (center_neigh)
                {
                    x = i + (ii - neib_rows / 2) * step_rows;
                    y = j + (jj - neib_cols / 2) * step_cols;
                }

                // Wrap the indices around the bounds of the input matrix if "wrap_mode" is set.
                if (wrap_mode)
                {
                    x = (x + rows) % rows;
                    y = (y + cols) % cols;
                }

                // Set the element in the output matrix
                if (x >= 0 && x < rows && y >= 0 && y < cols)
                {
                    // Set output matrix element i,j,ii,jj to the input matrix element x,y.
                    output[i * cols + j * neib_rows * neib_cols + ii * neib_cols + jj] = input[x * cols + y];
                }
                else
                {
                    // Set the element in the output matrix to 0 if the indices are outside the bounds of the input matrix.
                    output[i * cols + j * neib_rows * neib_cols + ii * neib_cols + jj] = 0;
                }
            }
        }
    }
}

///-----------------------------------------------------------------------------
///
/// gpu_Images2Neibs           A function that performs a sliding window operation on a matrix.
///                            This function is designed to be called from the host. It allocates
///                            memory on the GPU, copies the input matrix to the GPU, launches the
///                            sliding_window_kernel kernel function, copies the output matrix from the GPU
///                            and frees the memory on the GPU.
///
/// @param[in] input           A reference to the input matrix on the host. This is a 1D vector simulating a 2D matrix.
/// @param[in] input_shape     A pair containing the number of rows and columns in the input matrix.
/// @param[in] neib_shape      A pair containing the number of rows and columns in the neighbourhood.
/// @param[in] neib_step       A pair containing the number of rows and columns to step the neighbourhood for each iteration.
/// @param[in] wrap_mode       A flag indicating whether the neighbourhood should wrap around the input matrix.

std::vector<int> gpu_Images2Neibs(
    const std::vector<int> &input,
    const std::pair<int, int> &input_shape,
    const std::pair<int, int> &neib_shape,
    const std::pair<int, int> &neib_step,
    bool wrap_mode,
    bool center_neigh)
{
    // Determine the dimensions of the input matrix.
    const int rows = input_shape.first;
    const int cols = input_shape.second;

    // Check that the neighbourhood shape is valid.
    if (neib_shape.first > rows || neib_shape.second > cols)
    {
        throw std::invalid_argument("Neighbourhood shape must not be larger than the input matrix");
    }

    // Set the default step size to the neighbourhood shape.
    std::pair<int, int> step = neib_step;
    if (step.first == 0 && step.second == 0)
    {
        step = neib_shape;
    }

    int N = static_cast<int>(ceil(static_cast<float>(rows) / step.first));  // Number of rows in output matrix
    int M = static_cast<int>(ceil(static_cast<float>(cols) / step.second)); // Number of columns in output matrix
    int O = neib_shape.first;                                               // Number of rows in each patch
    int P = neib_shape.second;                                              // Number of columns in each patch

    // Create the output matrix. A 1D vector simulating a 4D vector with dimensions N x M x O x P.
    std::vector<int> output;

    // Allocate memory on the GPU for the input matrix.
    int *d_input, *d_output;

    // allocate device storage for the input matrix. The host (CPU) already has storage for the input.
    hipMalloc(&d_input, rows * cols * sizeof(int));
    output.resize(N * M * O * P);
    hipMalloc(&d_output, N * M * O * P * sizeof(int));

    // copy the input matrix to the GPU. Copy from the first element in the multi dim vector.
    hipMemcpy(d_input, input.data(), rows * cols * sizeof(int), hipMemcpyHostToDevice);

    // launch the kernel function on the GPU.
    int threadsPerBlock = 256;
    dim3 block(16, 16); // 256 threads per block. A standard value this can be increased on some GPU models.
    int noOfBlocks = cols * rows / 256;
    if ((cols * rows) % threadsPerBlock)
    {
        noOfBlocks++;
    }
    dim3 grid((cols + 16 - 1) / 16, (rows + 16 - 1) / 16);

    sliding_window_kernel<<<grid, block>>>(d_input, d_output, rows, cols, neib_shape.first, neib_shape.second, step.first, step.second, wrap_mode, center_neigh);

    // copy the output matrix back to the host. Copy to the pointer of the first element in the multi dim vector.
    hipMemcpy(output.data(), d_output, N * M * O * P * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}

// Function: main
int main(int argc, char *argv[])
{

    if (argc != 1)
    {
        std::cerr << "Usage: ./htm_flow" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    // Test 1: Check that a 2x2 patch is extracted from a 3x3 matrix
    // Create an input matrix for testing
    std::vector<std::vector<int>>
        input = {{1, 2, 3},
                 {4, 5, 6},
                 {7, 8, 9}};

    std::pair<int, int> input_shape = {input.size(), input[0].size()};
    // Set the neighbourhood shape and step size
    std::pair<int, int> neib_shape = {2, 2};
    std::pair<int, int> neib_step = {1, 1};
    bool wrap_mode = true;
    bool center_neigh = false;

    // We need to flatten the input matrix
    std::vector<int> flat_input = flattenVector(input);

    // Print the flat_input
    std::cout << "flat_input: " << std::endl;
    for (int i = 0; i < flat_input.size(); i++)
    {
        std::cout << flat_input[i] << ", ";
    }

    // Run the function and save the output
    std::vector<int> flat_output = gpu_Images2Neibs(flat_input, input_shape, neib_shape, neib_step, wrap_mode, center_neigh);

    // Print the flat output
    std::cout << "\nflat_output: " << std::endl;
    for (int i = 0; i < flat_output.size(); i++)
    {
        std::cout << flat_output[i] << ", ";
    }

    // Unflatten the output
    auto output = unflattenVector(flat_output, input_shape.first, input_shape.second, neib_shape.first, neib_shape.second);
}